#include "hip/hip_runtime.h"
extern "C"
void init(float *t_height_map,
float *w_height_map,
float *s_height_map,
int SIZE_X,
int SIZE_Y);

extern "C"
void run_hydro_erosion(int cycles,
float t_step,
float min_tilt_angle,
float SEDIMENT_CAP,
float DISSOLVE_CONST,
float DEPOSIT_CONST,
int SIZE_X,
int SIZE_Y,
float PIPE_LENGTH,
float ADJACENT_LENGTH,
float TIME_STEP,
float MIN_TILT_ANGLE);

extern "C"
void free_mem();

extern "C"
void procedural_rain(float *water_height_map, float *rain_map, int SIZE_X, int SIZE_Y);

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <iostream>
#include <algorithm>
#include <random>

// includes CUDA
#include <hip/hip_runtime.h>

using namespace std;

#define FLOW_RIGHT 0
#define FLOW_UP 1
#define FLOW_LEFT 2
#define FLOW_DOWN 3
#define X_VEL 0
#define Y_VEL 1
#define LEFT_CELL row, col - 1
#define RIGHT_CELL row, col + 1
#define ABOVE_CELL row - 1, col
#define BELOW_CELL row + 1, col

// CUDA API error checking macro
#define T 1024
#define M 1536
#define blockSize 1024
#define cudaCheck(error) \
  if (error != hipSuccess) { \
    printf("Fatal error: %s at %s:%d\n", \
      hipGetErrorString(error), \
      __FILE__, __LINE__); \
    exit(1); \
              }


__global__ void update_water_flow(float *water_height_map, float *water_flow_map, float *d_updated_water_flow_map, int SIZE_X, int SIZE_Y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int col = index % SIZE_X;
    int row = index / SIZE_X;

    index = row * (SIZE_X * 4) + col * 4;   // 3D index
#ifdef FIX
    if ((row >= SIZE_Y) || (col >= SIZE_X)) return;
#endif
    d_updated_water_flow_map[index + FLOW_RIGHT] = 0;
    d_updated_water_flow_map[index + FLOW_UP] = 0;
    d_updated_water_flow_map[index + FLOW_LEFT] = 0;
    d_updated_water_flow_map[index + FLOW_DOWN] = 0;

}

static float *terrain_height_map;
static float *water_height_map;
static float *sediment_height_map;

void init(float *t_height_map,
    float *w_height_map,
    float *s_height_map,
    int SIZE_X,
    int SIZE_Y)
{
    /* set vars HOST*/
    terrain_height_map = t_height_map;
    water_height_map = w_height_map;
    sediment_height_map = s_height_map;
}

void run_hydro_erosion(int cycles,
    float t_step,
    float min_tilt_angle,
    float SEDIMENT_CAP,
    float DISSOLVE_CONST,
    float DEPOSIT_CONST,
    int SIZE_X,
    int SIZE_Y,
    float PIPE_LENGTH,
    float ADJACENT_LENGTH,
    float TIME_STEP,
    float MIN_TILT_ANGLE)
{
    int numBlocks = (SIZE_X * SIZE_Y + (blockSize - 1)) / blockSize;
    int SIZE = SIZE_X * SIZE_Y * sizeof(float);

    float *d_terrain_height_map, *d_updated_terrain_height_map;
    float *d_water_height_map, *d_updated_water_height_map;
    float *d_sediment_height_map, *d_updated_sediment_height_map;

    float *d_suspended_sediment_level;
    float *d_updated_suspended_sediment_level;
    float *d_water_flow_map;
    float *d_updated_water_flow_map;
    float *d_prev_water_height_map;
    float *d_water_velocity_vec;
    float *d_rain_map;

    cudaCheck(hipMalloc(&d_water_height_map, SIZE));
    cudaCheck(hipMalloc(&d_updated_water_height_map, SIZE));
    cudaCheck(hipMalloc(&d_prev_water_height_map, SIZE));
    cudaCheck(hipMalloc(&d_water_flow_map, SIZE * 4));
    cudaCheck(hipMalloc(&d_updated_water_flow_map, SIZE * 4)); // changing this array also changes d_terrain_height_map
    cudaCheck(hipMalloc(&d_terrain_height_map, SIZE));
    cudaCheck(hipMalloc(&d_updated_terrain_height_map, SIZE));
    cudaCheck(hipMalloc(&d_sediment_height_map, SIZE));
    cudaCheck(hipMalloc(&d_updated_sediment_height_map, SIZE));
    cudaCheck(hipMalloc(&d_suspended_sediment_level, SIZE));
    cudaCheck(hipMalloc(&d_updated_suspended_sediment_level, SIZE));
    cudaCheck(hipMalloc(&d_rain_map, SIZE));
    cudaCheck(hipMalloc(&d_water_velocity_vec, SIZE * 2));

    cudaCheck(hipMemcpy(d_terrain_height_map, terrain_height_map, SIZE, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_water_height_map, water_height_map, SIZE, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_sediment_height_map, sediment_height_map, SIZE, hipMemcpyHostToDevice));

    cout << "init terrain_height_map" << endl;
    for (int i = 0; i < SIZE_X * SIZE_Y; i++) {
        cout << terrain_height_map[i] << ", ";
        if (i % SIZE_X == 0 && i != 0) cout << endl;
    }

    /* launch the kernel on the GPU */
    float *temp;
    while (cycles--) {
        update_water_flow << < numBlocks, blockSize >> >(d_water_height_map, d_water_flow_map, d_updated_water_flow_map, SIZE_X, SIZE_Y);
        temp = d_water_flow_map;
        d_water_flow_map = d_updated_water_flow_map;
        d_updated_water_flow_map = temp;
    }
    cudaCheck(hipMemcpy(terrain_height_map, d_terrain_height_map, SIZE, hipMemcpyDeviceToHost));


    cout << "updated terrain" << endl;
    for (int i = 0; i < SIZE_X * SIZE_Y; i++) {
        cout << terrain_height_map[i] << ", ";
        if (i % SIZE_X == 0 && i != 0) cout << endl;
    }
}
