
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std ;

# define DELLEXPORT extern "C" __declspec(dllexport)

__global__ void kernel(long* answer = 0){
    *answer = threadIdx.x + (blockIdx.x * blockDim.x);
}

DELLEXPORT void resoult(long* h_answer){

    long* d_answer = 0;
    
    hipMalloc(&d_answer, sizeof(long));

    kernel<<<10,1000>>>(d_answer);
    hipMemcpy(&h_answer, d_answer, sizeof(long), hipMemcpyDeviceToHost);
        hipFree(d_answer);
}
