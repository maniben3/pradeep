
#include <hip/hip_runtime.h>
#include <hipblas.h>

typedef float ftype;
extern "C"

void run(ftype *i1, ftype  *i2, ftype *o1, int d);

void run(ftype *i1, ftype  *i2, ftype *o1, int d){

	ftype *d_i1, *d_i2, *d_o1;
	int ds = d*d*sizeof(ftype);
	hipMalloc(&d_i1, ds);
	hipMalloc(&d_i2, ds);
	hipMalloc(&d_o1, ds);
	hipMemcpy(d_i1, i1, ds, hipMemcpyHostToDevice);
	hipMemcpy(d_i2, i2, ds, hipMemcpyHostToDevice);

	hipblasHandle_t h;
	hipblasCreate(&h);
	ftype alpha = 1.0;
	ftype beta = 0.0;
	hipblasSgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, d, d, d, &alpha, d_i1, d, d_i2, d, &beta, d_o1, d);
	hipMemcpy(o1, d_o1, ds, hipMemcpyDeviceToHost);
	
	hipFree(d_i1);
	hipFree(d_i2);
	hipFree(d_o1);
}
